#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <fstream>
#include <iostream>
#include <conio.h>

using namespace std;


__device__
int Min(int a, int b) { return a < b ? a : b; }

__global__ void RoyFloyd(int* mat, int k, int N) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < N && j < N) {
    if (mat[i*N + k] != -1 && mat[k*N + j] != -1) {
      if (mat[i*N+j] == -1) {
        mat[i*N+j] = mat[i*N + k] + mat[k*N +j];
      } else {
        mat[i*N+j] = Min(mat[i*N + k] + mat[k*N + j], mat[i*N+j]);
      }
    }
  }

}


int main(int argc, char **argv) {
  
  int thread_per_block = 512, i, j;
  int* cuda_mat;
  size_t N;

  ifstream f("date.in");
  f >> N;

  cout <<" n = "  << N<<endl;


  int size = sizeof(int)* N * N;

  int *mat = (int*)malloc(sizeof(int)*N*N);

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      int v;
      f >> v;
      mat[i * N + j] = v;
    }
  }

  int *result = (int*)malloc(sizeof(int)*N*N);

  clock_t t;
  t = clock();

  hipMalloc((void**)&cuda_mat, size);
  hipMemcpy(cuda_mat, mat, size, hipMemcpyHostToDevice);
  int num_block = ceil((float)(N*N / (thread_per_block)));

  for (int k = 0; k < N; ++k) {
    RoyFloyd << <num_block, (thread_per_block) >> >(cuda_mat, k, N);
  }
  hipMemcpy(result, cuda_mat, size, hipMemcpyDeviceToHost);
  hipFree(cuda_mat);
  t = clock() - t;

  cout<<" execution time: "<<((double)t)/CLOCKS_PER_SEC<<endl;

  getchar();
}
